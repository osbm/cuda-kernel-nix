
#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
        int run_version, driver_version;
        std::cout<< "Return Code Runtime Version: ";
        std::cout<<hipRuntimeGetVersion(&run_version);
        std::cout<<"\nReturn Code Driver Version: ";
        std::cout<<hipDriverGetVersion(&driver_version);
        std::cout << "\nRuntime Version: ";
        std::cout << run_version;
        std::cout << "\n Driver Version: ";
        std::cout << driver_version;
        return 0;
}

