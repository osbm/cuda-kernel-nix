#include <stdio.h>
#include "hip/hip_runtime.h"
extern "C"

void checkGpuMem() {
    float free_m, total_m, used_m;
    size_t free_t,total_t;
    hipMemGetInfo(&free_t,&total_t);
    free_m =(uint)free_t/1048576.0 ;
    total_m=(uint)total_t/1048576.0;
    used_m=total_m-free_m;
    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",used_m,free_m,total_m);
}