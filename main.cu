
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>


int main(){

    size_t mf, ma;
    hipMemGetInfo(&mf, &ma);
    std::cout << "Free memory (gb): " << mf/1024/1024/1024 << std::endl;
    std::cout << "Total memory (gb): " << ma/1024/1024/1024 << std::endl;

    return 0;
}
