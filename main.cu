
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>


int main(){

    size_t mf, ma;
    hipMemGetInfo(&mf, &ma);
    std::cout << "Free memory (mb): " << mf/1024/1024 << std::endl;
    std::cout << "Total memory (mb): " << ma/1024/1024 << std::endl;

    return 0;
}
